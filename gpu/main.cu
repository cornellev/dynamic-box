#include "hip/hip_runtime.h"
#include <iostream>
#include <filesystem>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <Eigen/Core>
#include <Eigen/Geometry>

static const int DEVICE_ID = 0;

static const Eigen::Matrix3f K_l{{699.41, 0.0, 652.8}, {0.0, 699.365, 358.133}, {0.0, 0.0, 1.0}};
static const Eigen::Matrix3f K_r{{697.635, 0.0, 671.665}, {0.0, 697.63, 354.611}, {0.0, 0.0, 1.0}};

static const Eigen::Matrix3f T{
    {0.0, 0.467919, 0.0458908}, {-0.467919, 0.0, -119.905}, {-0.0458908, 119.905, 0.0}};
static const Eigen::Vector3f R_Rodrigues = Eigen::Vector3f{0.00239722, 0.00697667, -0.0021326};
static const Eigen::Matrix3f R =
    Eigen::AngleAxisf(R_Rodrigues.norm(), R_Rodrigues.normalized()).toRotationMatrix();

static const Eigen::Matrix3f E = R * T;
static const Eigen::Matrix3f F = K_r.transpose().inverse() * E * K_l.inverse();

static const int32_t SAD_KERNEL_SIZE = 11;
static_assert(SAD_KERNEL_SIZE % 2 == 1, "SAD kernel size must be odd");

static const int32_t MIN_DISPARITY = 0;
static const int32_t MAX_DISPARITY = 128;

static const float MIN_DISPLAY_DEPTH = 0.0;
static const float MAX_DISPLAY_DEPTH = 10000.0;

__global__ void calculateDepth(const uchar3* left, const uchar3* right, float* depth, int32_t rows,
    int32_t cols, Eigen::Matrix3f F) {
    extern __shared__ uchar3 shared[];

    int32_t id = blockIdx.x * blockDim.x + threadIdx.x;

    int32_t row = id / cols;
    int32_t col = id % cols;

    for (int32_t i = 0; i < SAD_KERNEL_SIZE; i++) {
        int32_t s_idx = blockDim.x * i + threadIdx.x;
        int32_t row_offset = i - SAD_KERNEL_SIZE / 2;

        int32_t r_row = (int32_t)row + row_offset;
        if (r_row < 0 || r_row >= rows) {
            shared[s_idx] = {0, 0, 0};
        } else {
            shared[s_idx] = right[r_row * cols + col];
        }
    }

    __syncthreads();

    Eigen::Vector3f l_pixel{col, row, 1.0};
    Eigen::Vector3f epipolar_line = F * l_pixel;
    int32_t r_row = round(-epipolar_line(2) / epipolar_line(1));

    int32_t search_start = max(0, col - MAX_DISPARITY);
    int32_t search_end = max(0, min(col, col - MIN_DISPARITY));

    uint32_t min_sad = UINT32_MAX;
    int32_t best_r_col = search_start;

    uint64_t shared_count = 0;
    uint64_t global_count = 0;

    for (int32_t r_col = search_start; r_col <= search_end; r_col++) {
        uint32_t sad = 0;

        for (int32_t i = 0; i < SAD_KERNEL_SIZE; i++) {
            for (int32_t j = 0; j < SAD_KERNEL_SIZE; j++) {
                int32_t l_ker_row = row + i - SAD_KERNEL_SIZE / 2;
                int32_t l_ker_col = col + j - SAD_KERNEL_SIZE / 2;
                int32_t r_ker_row = r_row + i - SAD_KERNEL_SIZE / 2;
                int32_t r_ker_col = r_col + j - SAD_KERNEL_SIZE / 2;

                if (l_ker_row < 0 || l_ker_row >= rows || l_ker_col < 0 || l_ker_col >= cols
                    || r_ker_row < 0 || r_ker_row >= rows || r_ker_col < 0 || r_ker_col >= cols) {
                    continue;
                }

                int32_t s_r_i = i;
                int32_t s_r_j = threadIdx.x - (col - r_ker_col);
                int32_t s_idx = blockDim.x * s_r_i + s_r_j;

                uchar3 l_pixel = left[l_ker_row * cols + l_ker_col];
                uchar3 r_pixel;
                if (s_r_j >= 0 && s_r_j < blockDim.x) {
                    r_pixel = shared[s_idx];
                    shared_count++;
                } else {
                    r_pixel = right[r_ker_row * cols + r_ker_col];
                    global_count++;
                }

                sad += abs(l_pixel.x - r_pixel.x) + abs(l_pixel.y - r_pixel.y)
                       + abs(l_pixel.z - r_pixel.z);
            }
        }

        // TODO: branchless?
        if (sad < min_sad) {
            min_sad = sad;
            best_r_col = r_col;
        }
    }

    float disparity = col - best_r_col;
    depth[id] = 119.905 * 699.41 / disparity;  // mm
}

int main() {
    hipSetDevice(DEVICE_ID);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    cv::Mat3b left = cv::imread("../left/left1.png", cv::IMREAD_COLOR);
    cv::Mat3b right = cv::imread("../right/right1.png", cv::IMREAD_COLOR);
    assert(left.rows == right.rows && left.cols == right.cols);

    auto start = std::chrono::high_resolution_clock::now();
    cv::Mat1f depth(left.rows, left.cols);

    int32_t stereo_bytes = left.total() * left.elemSize();
    int32_t depth_bytes = depth.total() * depth.elemSize();

    uchar3* d_left;
    uchar3* d_right;
    float* d_depth;
    hipMalloc(&d_left, stereo_bytes);
    hipMalloc(&d_right, stereo_bytes);
    hipMalloc(&d_depth, depth_bytes);

    hipMemcpy(d_left, left.data, stereo_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_right, right.data, stereo_bytes, hipMemcpyHostToDevice);

    int32_t threads = left.rows * left.cols;
    int32_t threads_per_block = 1024;
    int32_t blocks_per_grid = (threads + threads_per_block - 1) / threads_per_block;
    int32_t shared_mem = threads_per_block * SAD_KERNEL_SIZE * sizeof(uchar3);

    std::cout << "Using " << threads_per_block << " threads per block and " << blocks_per_grid
              << " blocks\n";
    std::cout << "Using " << shared_mem / 1024.0 << "kiB of shared memory per block\n";

    auto start_kernel = std::chrono::high_resolution_clock::now();
    calculateDepth<<<blocks_per_grid, threads_per_block, shared_mem>>>(d_left, d_right, d_depth,
        left.rows, left.cols, F);

    hipDeviceSynchronize();
    auto end_kernel = std::chrono::high_resolution_clock::now();
    std::cout
        << "Kernel execution took "
        << std::chrono::duration_cast<std::chrono::milliseconds>(end_kernel - start_kernel).count()
        << " ms" << std::endl;

    hipMemcpy(depth.data, d_depth, depth_bytes, hipMemcpyDeviceToHost);

    cv::Mat1b depth_8u;
    depth.convertTo(depth_8u, CV_8U, 255.0 / (MAX_DISPLAY_DEPTH - MIN_DISPLAY_DEPTH),
        -255.0 * MIN_DISPLAY_DEPTH / (MAX_DISPLAY_DEPTH - MIN_DISPLAY_DEPTH));

    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Depth calculation took "
              << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms"
              << std::endl;

    start = std::chrono::high_resolution_clock::now();

    cv::Ptr<cv::StereoBM> stereo = cv::StereoBM::create();
    stereo->setBlockSize(SAD_KERNEL_SIZE);
    stereo->setMinDisparity(MIN_DISPARITY);
    stereo->setNumDisparities(MAX_DISPARITY - MIN_DISPARITY);

    cv::Mat1b left_gray;
    cv::cvtColor(left, left_gray, cv::COLOR_BGR2GRAY);
    cv::Mat1b right_gray;
    cv::cvtColor(right, right_gray, cv::COLOR_BGR2GRAY);

    cv::Mat disparity_cv;
    stereo->compute(left_gray, right_gray, disparity_cv);
    cv::Mat1f depth_cv;
    disparity_cv.convertTo(depth_cv, CV_32F);
    depth_cv = 119.905 * 699.41 / (depth_cv / 16.0);
    cv::Mat1b depth_cv_8u;
    depth_cv.convertTo(depth_cv_8u, CV_8U, 255.0 / (MAX_DISPLAY_DEPTH - MIN_DISPLAY_DEPTH),
        -255.0 * MIN_DISPLAY_DEPTH / (MAX_DISPLAY_DEPTH - MIN_DISPLAY_DEPTH));

    end = std::chrono::high_resolution_clock::now();
    std::cout << "OpenCV depth calculation took "
              << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms"
              << std::endl;

    std::filesystem::create_directory("output");
    cv::imwrite("output/depth.png", depth_8u);
    cv::imwrite("output/depth_cv.png", depth_cv_8u);

    cv::imshow("Depth", depth_8u);
    cv::waitKey(0);

    hipFree(d_left);
    hipFree(d_right);
    hipFree(d_depth);

    return 0;
}